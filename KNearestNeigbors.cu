#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>
#include <fstream>


#define BLOCK_SIZE 16

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

using idx_t = std::uint32_t;
using val_t = float;
using ptr_t = std::uintptr_t;

typedef struct csr_t {
  idx_t nrows; // number of rows
  idx_t ncols; // number of rows
  idx_t * ind; // column ids
  val_t * val; // values
  ptr_t * ptr; // pointers (start of row in ind/val)

  csr_t()
  {
    nrows = ncols = 0;
    ind = nullptr;
    val = nullptr;
    ptr = nullptr;
  }

  void reserve(const idx_t nrows, const ptr_t nnz)
  {
    if(nrows > this->nrows){
      if(ptr){
        ptr = (ptr_t*) realloc(ptr, sizeof(ptr_t) * (nrows+1));
      } else {
        ptr = (ptr_t*) malloc(sizeof(ptr_t) * (nrows+1));
        ptr[0] = 0;
      }
      if(!ptr){
        throw std::runtime_error("Could not allocate ptr array.");
      }
    }
    if(ind){
      ind = (idx_t*) realloc(ind, sizeof(idx_t) * nnz);
    } else {
      ind = (idx_t*) malloc(sizeof(idx_t) * nnz);
    }
    if(!ind){
      throw std::runtime_error("Could not allocate ind array.");
    }
    if(val){
      val = (val_t*) realloc(val, sizeof(val_t) * nnz);
    } else {
      val = (val_t*) malloc(sizeof(val_t) * nnz);
    }
    if(!val){
      throw std::runtime_error("Could not allocate val array.");
    }
    this->nrows = nrows;
  }

  void read(const std::string &filename)
  {
    FILE * infile = fopen(filename.c_str(), "r");
    char * line = NULL;
    size_t n, nr, nnz;
    char *head;
    char *tail;
    idx_t cid;
    double dval;
    
    if (!infile) {
      throw std::runtime_error("Could not open CLU file\n");
    }
    if(getline (&line, &n, infile) < 0){
      throw std::runtime_error("Could not read first line from CLU file\n");
    }
    //read matriz size info
    size_t rnrows, rncols, rnnz;
    sscanf(line, "%zu %zu %zu", &rnrows, &rncols, &rnnz);

    //allocate space
    this->reserve(rnrows, rnnz);
    ncols = rncols;
    
    //read in rowval, rowind, rowptr
    this->ptr[0]= 0;
    nnz = 0;
    nr = 0;

    while(getline(&line, &n, infile) != -1){
      head = line;
      while (1) {
        cid = (idx_t) strtol(head, &tail, 0);
        if (tail == head)
          break;
        head = tail;

        if(cid <= 0){
          throw std::runtime_error("Invalid column ID while reading CLUTO matrix\n");
        }
        this->ind[nnz] = cid - 1; //csr/clu files have 1-index based column IDs and our matrix is 0-based.
        dval = strtod(head, &tail);
        head = tail;
        this->val[nnz++] = dval;
      }
      this->ptr[nr+1] = nnz;
      nr++;
    }
    assert(nr == rnrows);
    free(line);
    fclose(infile);
  }

  static csr_t * from_CLUTO(const std::string &filename)
  {
    auto mat = new csr_t();
    mat->read(filename);
    return mat;
  }

  void write(const std::string output_fpath, const bool header=false)
  {
    std::fstream resfile;
    resfile.open(output_fpath, std::ios::out);
    if(!resfile){
      throw std::runtime_error("Could not open output file for writing.");
    }
    if(header){
      resfile << nrows << " " << ncols << " " << ptr[nrows] << std::endl;
    }
    for(idx_t i=0; i < nrows; ++i){
      for(ptr_t j=ptr[i]; j < ptr[i+1]; ++j){
        resfile << ind[j] << " " << val[j];
        if(j+1 < ptr[i+1]){
          resfile << " ";
        }
      }
      resfile << std::endl;
    }
    resfile.close();
  }

  __global__ void normalize(int norm=2)
  {
    val_t sum;
    //for (idx_t i = 0; i < nrows; i++) { // each row
    int i = threadIdx.x;
    sum = 0;
    for (ptr_t j = ptr[i]; j < ptr[i + 1]; j++) { // each value in row
        if (norm == 2) {
        sum += val[j] * val[j];
        } else if (norm == 1) {
        sum += val[j] > 0 ? val[j] : -val[j];
        } else {
        throw std::runtime_error("Norm must be 1 or 2.");
        }
    }
    if (sum > 0) {
        if (norm == 2) {
        sum = (double) 1.0 / sqrt(sum);
        } else {
        sum = (double) 1.0 / sum;
        }
        for (ptr_t j = ptr[i]; j < ptr[i + 1]; j++) {
        val[j] *= sum;
        }
    }
  }

  ~csr_t()
  {
    if(ind){
      free(ind);
    }
    if(val){
      free(val);
    }
    if(ptr){
      free(ptr);
    }
  }
} csr_t;